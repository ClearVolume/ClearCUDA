#include "hip/hip_runtime.h"

#include "hip/hip_vector_types.h"

extern "C" __global__ void funcname(int length, float *a, float *b, float *c)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
	if (x < length)
	{
		c[x] = a[x] + b[x];
	}
}

  