#include "hip/hip_runtime.h"
#define N (1024*1024)
#define M (1000000)

#include "hip/hip_vector_types.h"

extern "C" __global__ void funcname(float *a, float *b, float *c)
{
	int id = blockIdx.x;
	if (id < N)
	{
		c[id] = a[id] + b[id];
	}
}

  